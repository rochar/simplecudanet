#include "hip/hip_runtime.h"
#include "AddVectorsKernel.cuh"

#include "hip/hip_runtime.h"
#include ""
__global__ void AddVectorsKernel(int n, int* v1, int* v2, int* result)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < n)
	{
		result[index] = v1[index] + v2[index];
	}
}


hipError_t RunAddVectorsKernel(int elementsToAdd, int* device_v1, int* device_v2, int* device_result)
{
	int blockSize = 256;
	int numBlocks = (elementsToAdd + blockSize - 1) / blockSize;
	
	AddVectorsKernel <<<numBlocks, blockSize >>> (elementsToAdd, device_v1, device_v2, device_result);

	return hipDeviceSynchronize();
}